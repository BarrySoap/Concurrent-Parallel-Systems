#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <vector>

using namespace std;

//constexpr size_t ELEMENTS = 2048;

__global__ void vecadd(const int *A, const int *B, int *C)
{
	// Get block index
	unsigned int block_idx = blockIdx.x;
	// Get thread index
	unsigned int thread_idx = threadIdx.x;
	// Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	// Get the thread's unique ID - (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	// Add corresponding locations of A and B and store in C
	C[idx] = A[idx] + B[idx];
}

int main(int argc, char **argv)
{
	/* Example 1: GPU Specifications */
	// Get number of devices on system
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	cout << "Number of devices: " << deviceCount << endl;
	for (int i = 0; i < deviceCount; ++i)
	{
		// Get properties for device
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);

		cout << "Device " << i << endl;
		cout << "Name " << deviceProp.name << endl;
		cout << "Revision " << deviceProp.major << "." << deviceProp.minor << endl;
		cout << "Memory " << deviceProp.totalGlobalMem / 1024 / 1024 << "MB" << endl;
		cout << "Warp Size " << deviceProp.warpSize << endl;
		cout << "Clock " << deviceProp.clockRate << endl;
		cout << "Multiprocessors " << deviceProp.multiProcessorCount << endl;
	}
	return 0;
	/********************************/

	/* Example 2: Vector Addition /
	// Create host memory
	auto data_size = sizeof(int) * ELEMENTS;
	vector<int> A(ELEMENTS);    // Input aray
	vector<int> B(ELEMENTS);    // Input array
	vector<int> C(ELEMENTS);    // Output array

	// Initialise input data
	for (unsigned int i = 0; i < ELEMENTS; ++i)
		A[i] = B[i] = i;

	// Declare buffers
	int *buffer_A, *buffer_B, *buffer_C;

	// Initialise buffers
	hipMalloc((void**)&buffer_A, data_size);
	hipMalloc((void**)&buffer_B, data_size);
	hipMalloc((void**)&buffer_C, data_size);

	// Write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_B, &B[0], data_size, hipMemcpyHostToDevice);

	// Run kernel with one thread for each element
	// First value is number of blocks, second is threads per block.  Max 1024 threads per block
	vecadd << <ELEMENTS / 1024, 1024 >> > (buffer_A, buffer_B, buffer_C);

	// Wait for kernel to complete
	hipDeviceSynchronize();

	// Read output buffer back to the host
	hipMemcpy(&C[0], buffer_C, data_size, hipMemcpyDeviceToHost);

	// Clean up resources
	hipFree(buffer_A);
	hipFree(buffer_B);
	hipFree(buffer_C);

	// Test that the results are correct
	for (int i = 0; i < 2048; ++i)
		if (C[i] != i + i)
			cout << "Error: " << i << endl;

	cout << "Finished" << endl;

	return 0;
	/********************************/
}